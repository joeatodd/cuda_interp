#include "hip/hip_runtime.h"
/* 3D NetCDF interpolation using CUDA

   Goal: Enable accelerated regridding of 3D data from NetCDF format

   Kernel: Trilinear interpolation w/ stored weights for repeat interp

   Challenges:

     - Off-axis grids
     - Calculation of block memory footprint
     - Negotiation of per-block parallel read

   Notes:

   For each point in dest_arr, only need to store id of single
   point in src_arr corresponding to bottom-left-near corner
   (min x,y,z) and <8> weights.

   If memory limited, could recompute 8 weights on-the-fly &
   only store <3> local coords.

   Each block corresponds to an (nx, ny, nz) subset of the dest_arr,
   and a corresponding subset of the src_arr (inc. ghost).

   Metis would take care of this, but let's do it manually as a
   learning exercise. Most efficient (minimize edge-cut)
*/

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "data.h"
#include "kernels.h"
#include "types.h"
#include "config.h"

using std::vector;
using std::cout;
using std::endl;

int main(int argc, char* argv[]){

  config_t config = getCfg(argv[1]);

  vector<std::string> dim_names = {"x","y","z"};
//   vector<std::string> var_names = {"temperature"};
//   // fields_t fields(var_names.size());
  vector<field_t<double>> fields(config.variables.size());
  coords_t coords(3);

  int retval = getNetcdfData(config.input_filename, config.dim_names, config.variables, coords, fields);
  if(retval){cout << "NetCDF Error" << endl; return retval;}

  // try the max function
  // ***********************************************
#if 0
  double *dfield, *dmax, *dtest;
  int nvals = fields[0].size();
  int blocksize = 512;
  int nblocks = (nvals + blocksize - 1) / blocksize;

  dtest = (double*)malloc(nvals*sizeof(double));
  gpuErrchk(hipMalloc((void **)&dfield, nvals*sizeof(double)));
  gpuErrchk(hipMalloc((void **)&dmax, nvals*sizeof(double))); // TODO too big!

  gpuErrchk(hipMemcpy(dfield, &fields[0][0],nvals*sizeof(double), hipMemcpyHostToDevice));

  // Repeated calls to kernel w/ block-level reduction
  int cnt = nvals;
  while(cnt > 1){
    cout << "Operating on cnt: " << cnt << endl;
    maxVal<<<nblocks, blocksize, blocksize*sizeof(double)>>>(cnt, dfield, dmax);
    gpuErrchk(hipMemcpy(dfield, dmax, nvals*sizeof(double) ,hipMemcpyDeviceToDevice));
    cnt = (cnt + blocksize - 1) / blocksize;
  }

  double maxVal;
  gpuErrchk(hipMemcpy(&maxVal, dmax, sizeof(double), hipMemcpyDeviceToHost));

  cout << "Max val from gpu: " << maxVal << endl;
  auto maxy = std::max_element(std::begin(fields[0]), std::end(fields[0]));
  cout << "Max val from cpu:" << *maxy << endl;

  // It works!
  // ***********************************************
#endif

  // Generate specification of new grid for interp (should be read in)
  gridspec_t gridSpecOut, gridSpecIn;
  gridSpecOut = config.gridspec_out;
  gridSpecIn = getNetcdfGrid(coords);

  // Copy fields to new vector & delete all values
  // Note that we thus ensure that dimension order is consistent
  // between input & output (fields[i].dim_order)
  vector<field_t<double>> interped_fields(fields.size()); // TODO proper copy constructor?
  for (unsigned int i = 0; i < fields.size(); i++){
    interped_fields[i].name = fields[i].name;
    interped_fields[i].dims = fields[i].dims;
    interped_fields[i].dim_order = fields[i].dim_order;
  }

  // CPU interp for benchmarking
  bool test = false;
  if(test){
    // Compute the coords of each point in new grid in the 'netcdf index' space
    coords_t localCoords = gridToGrid3D(gridSpecIn, gridSpecOut);
    cpuTrilinInterp(localCoords, fields, gridSpecIn, gridSpecOut, interped_fields);
  }

  gpuTrilinInterp(gridSpecIn, gridSpecOut, fields, interped_fields);

  retval = writeNetcdfData(config.output_filename, gridSpecOut, config.dim_names, interped_fields);
  if(retval){cout << "NetCDF Error" << endl; return retval;}

  return 0;

}
